#include "hip/hip_runtime.h"

// #include <gpu_particle_filter/gpu_particle_filter.h>
#include <gpu_particle_filter/particle_filter_kernel.h>

struct __align__(16) Points{
    float x;
    float y;

    __device__
        float magnitude(void) {
        return x * x + y *y;
    }
};

__global__
void hello(char *a, int *b) {
    a[threadIdx.x] += b[threadIdx.x];

}

void test_cuda(char* a, int* b) {

    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);

    printf("%s", a);

    hipMalloc((void**)&ad, csize);
    hipMalloc((void**)&bd, isize);
    hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, isize, hipMemcpyHostToDevice);

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy(a, ad, csize, hipMemcpyDeviceToHost);
    hipFree(ad);
    hipFree(bd);
}

template<typename T>
__device__ __forceinline__
T cuFloor(const T x) {
    return static_cast<T>(std::floor(x));
}


__global__
void boxFilterGPU(char *pixels, const int fsize) {
    int t_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int t_idy = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = t_idx + t_idy * blockDim.x * gridDim.x;

    if (t_idx > fsize && t_idy > fsize &&
        t_idx < 640 - fsize && t_idy < 480 - fsize) {
        int val = 0;
        int icounter = 0;
        for (int i = -fsize; i < fsize; i++) {
            for (int j = -fsize; j < fsize; j++) {
                int idx = (t_idx - j) + (t_idy - i) * blockDim.x * gridDim.x;
                val += pixels[idx];
                icounter++;
            }
        }
        pixels[offset] = val/icounter;
    }
}

void boxFilter(cv::Mat &image, const int size) {
    // cv::cvtColor(image, image, CV_BGR2GRAY);
    int lenght = static_cast<int>(image.rows * image.cols) * sizeof(char);
    char *pixels = (char*)malloc(lenght);
    for (int i = 0; i < image.rows; i++) {
        for (int j = 0; j < image.cols; j++) {
            int index = j + (i * image.cols);
            pixels[index] = image.at<uchar>(i, j);
        }
    }
    char *d_pixels;
    hipMalloc((void**)&d_pixels, lenght);
    hipMemcpy(d_pixels, pixels, lenght, hipMemcpyHostToDevice);
    
    dim3 dim_thread(blocksize, blocksize);
    dim3 dim_block(static_cast<int>(image.cols)/dim_thread.x,
                   static_cast<int>(image.rows)/dim_thread.y);

    int b_start = static_cast<int>((float)size/2.0f);
    boxFilterGPU<<<dim_block, dim_thread>>>(d_pixels, b_start);

    char *download_pixels = (char*)malloc(lenght);
    hipMemcpy(download_pixels, d_pixels, lenght, hipMemcpyDeviceToHost);

    int stride = image.cols;
    int j = 0;
    int k = 0;
    for (int i = 0; i < lenght; i++) {
        if (i == stride) {
            j++;
            k = 0;
            stride += image.cols;
        }
        image.at<uchar>(j, k++) = download_pixels[i];
    }

    cv::imshow("image", image);
    cv::waitKey(0);
}
