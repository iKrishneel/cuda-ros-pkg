#include "hip/hip_runtime.h"

// #include <gpu_particle_filter/gpu_particle_filter.h>
#include <gpu_particle_filter/particle_filter_kernel.h>


__global__
void hello(char *a, int *b)
{

    a[threadIdx.x] += b[threadIdx.x];
}

void test_cuda(char* a, int* b) 
{

    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);

    printf("%s", a);

    hipMalloc( (void**)&ad, csize );
    hipMalloc( (void**)&bd, isize );
    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    hello<<<dimGrid, dimBlock>>>(ad, bd);
    hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );
    hipFree( ad );
    hipFree( bd );
}

